
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace std;

__global__ void vector_addition(int *A, int *B, size_t n) {
	A[threadIdx.x] += B[threadIdx.x];
}

int main() {
	const int n = 128;    int A[n] = {0};
	int B[n] = {0};

	for (int i = 0; i < n; ++i) A[i] = i, B[i] = n - i;
	
	int *Ad, *Bd;
	hipMalloc((void **)&Ad, sizeof(int)*n);
	hipMalloc((void **)&Bd, sizeof(int)*n);

	hipMemcpy(Ad, A, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(Bd, B, sizeof(int)*n, hipMemcpyHostToDevice);

	vector_addition<<<1, n>>>(Ad, Bd, n);
	hipMemcpy(A, Ad, sizeof(int)*n, hipMemcpyDeviceToHost);

	for (int i = 0; i < n; ++i) printf("%2d ", A[i]);
	printf("\n");
	
	hipFree(Ad);
	hipFree(Bd);
	
	return 0;
}
