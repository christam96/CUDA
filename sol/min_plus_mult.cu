#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <fstream>
#include <climits>
#include <algorithm>
#include <cmath>
#include <chrono>
#include <bits/stdc++.h> 
#include <math.h>

using namespace std;

void printMatrix(int *matrix, int matrixWidth) {
	for (int i = 0; i < matrixWidth*matrixWidth; i++) {
		if (i % matrixWidth == 0) {
			cout << endl;
		}

		cout << matrix[i] << " ";
	}
}

bool equivChecker(int *ResultMatrixMatrix, int *expectedMatrix, int matrixSize) {
	for (int k = 0; k < matrixSize; k++) {
		if (expectedMatrix[k] != ResultMatrixMatrix[k]) {
			return false;
		}
	}
	return true;
}

// function to evaluate logarithm base-2
int calculateLog(int d) 
{ 
	int x = log2(d);
	printf("Log %d is %d", d, x);
	return x; 
} 

__global__ void min_plus_kernel_cache_first(int *MatrixA, int *MatrixB, int *ResultMatrix, int matrixWidth) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ int sharedData[];
	sharedData[index] = MatrixA[index];
	__syncthreads();

	int resVal = INT_MAX;

	int col = index % matrixWidth;
	int row = index/matrixWidth;

	for (int k = 0; k < matrixWidth; k++) {
		int firstNum = sharedData[row*matrixWidth + k];
		int secondNum = MatrixB[k*matrixWidth + col];
			
		resVal = min(resVal, firstNum + secondNum);
	}
	
	ResultMatrix[index] = resVal;
}

__global__ void min_plus_kernel_cache_both(int *MatrixA, int *MatrixB, int *ResultMatrix, int matrixWidth) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = matrixWidth * matrixWidth;

	extern __shared__ int sharedData[];
	sharedData[index] = MatrixA[index];
	sharedData[index + offset] = MatrixB[index];
	__syncthreads();

	int resVal = INT_MAX;

	int col = index % matrixWidth;
	int row = index/matrixWidth;
	
	//each thread computes the correct ResultMatrix for a given index in the 2D array
	for (int k = 0; k < matrixWidth; k++) {
		int firstNum = sharedData[row*matrixWidth + k];
		int secondNum = sharedData[k*matrixWidth + col + offset];
		//int firstNum = sharedData[k];
		//int secondNum = MatrixB[k*matrixWidth + col];
			
		resVal = min(resVal, firstNum + secondNum);
	}
	
	ResultMatrix[index] = resVal;
}

__global__ void min_plus(int *MatrixA, int *MatrixB, int *ResultMatrix, int matrixWidth) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int rowNumber = blockIdx.x/(matrixWidth/blockDim.x);
	int firstIndexInRow = rowNumber*matrixWidth;
	
	ResultMatrix[index] = 1;

	extern __shared__ int sharedData[];
	int numberOfIndicesToLoad = matrixWidth/blockDim.x;
	if (matrixWidth % blockDim.x > threadIdx.x) {
		numberOfIndicesToLoad++;
	}

	for (int i = 0; i < numberOfIndicesToLoad; i++) {
		sharedData[threadIdx.x + i*1024] = MatrixA[firstIndexInRow + threadIdx.x + 1024*i];
	}

	__syncthreads();

	int resVal = INT_MAX;

	int col = index % matrixWidth;

	//each thread computes the correct ResultMatrix for a given index in the 2D array
	for (int k = 0; k < matrixWidth; k++) {
		int firstNum = sharedData[k];
		int secondNum = MatrixB[k*matrixWidth + col];
		
		resVal = min(resVal, firstNum + secondNum);
	}

	ResultMatrix[index] = resVal;
}

void min_plus_serial(int *MatrixA, int *MatrixB, int *ResultMatrix, int matrixWidth) {
	int numberOfEntries = matrixWidth * matrixWidth;
	for (int i = 0; i < numberOfEntries; i++) {
		ResultMatrix[i] = INT_MAX;
	}

	for (int row = 0; row < matrixWidth; row++) {
		for (int col = 0; col < matrixWidth; col++) {
			for (int k = 0; k < matrixWidth; k++) {
				int index = row*matrixWidth + col;
				ResultMatrix[index] = min(ResultMatrix[index], MatrixA[row*matrixWidth + k] + MatrixB[k*matrixWidth + col]);
			}
		}
	}
}

void implementAlgorithm(int argc, char *argv[]) {
	for (int i = 1; i < argc; i++) {
		int matrixWidth;

		//first number in file is matrix width
		ifstream myfile;
		myfile.open(argv[i]);
		myfile >> matrixWidth;

		//load first input matrix
		int sizeOfMatrix = matrixWidth*matrixWidth;
		int* MatrixA = (int*) malloc(sizeOfMatrix*sizeof(int));
		int* MatrixB = (int*) malloc(sizeOfMatrix*sizeof(int));
		int* ResultMatrix = (int*) malloc(sizeOfMatrix*sizeof(int));
		int* serialResultMatrix = (int*) malloc(sizeOfMatrix*sizeof(int));
		for (int j = 0; j < sizeOfMatrix; j++) {
			myfile >> MatrixA[j];
			ResultMatrix[j] = INT_MAX;
			serialResultMatrix[j] = INT_MAX;
		}	

		for (int j = 0; j < sizeOfMatrix; j++) {
			myfile >> MatrixB[j];
		}

		cout << "Matrix 1" << endl;
		printMatrix(MatrixA, matrixWidth);
		
		cout << endl << "Matrix 2" << endl;
		printMatrix(MatrixB, matrixWidth);
		

		//load expected ResultMatrix
		int* expected = (int*) malloc(sizeOfMatrix*sizeof(int));
		for (int j = 0; j < sizeOfMatrix; j++) {
			myfile >> expected[j];
		}

		int h = calculateLog(matrixWidth);

		cout << endl << "Expected" << endl;
		printMatrix(expected, matrixWidth);

		int* cudaMatrixA;
		int* cudaMatrixB;
		int* cudaResultMatrix;
		hipMalloc((void **) &cudaMatrixA, sizeof(int) * sizeOfMatrix);
		hipMalloc((void **) &cudaMatrixB, sizeof(int) * sizeOfMatrix);
		hipMalloc((void **) &cudaResultMatrix, sizeof(int) * sizeOfMatrix);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		
		hipMemcpy(cudaMatrixA, MatrixA, sizeof(int)*sizeOfMatrix, hipMemcpyHostToDevice);
		hipMemcpy(cudaMatrixB, MatrixB, sizeof(int)*sizeOfMatrix, hipMemcpyHostToDevice);
		hipMemcpy(cudaResultMatrix, ResultMatrix, sizeof(int)*sizeOfMatrix, hipMemcpyHostToDevice);

		//cout << endl << "ResultMatrix" << endl;
		//printMatrix(ResultMatrix, matrixWidth);

		// ALTERNATIVE KERNEL CALL
		// for (int i = 0; i < h; i++) {
		// 	if (i < h-1) {
		// 		if (matrixWidth < 128) {
		// 			int numberOfThreadBlocks = ceil(sizeOfMatrix/1024.0);
		// 			int numberOfThreads = min(1024, sizeOfMatrix);
		
		// 			if (matrixWidth < 64) {
		// 				int sharedMemorySize = sizeOfMatrix*2*sizeof(int);
		// 				hipEventRecord(start);
		// 				cout << "CACHING BOTH MATRICES:" << endl;
		// 				min_plus_kernel_cache_both<<<numberOfThreadBlocks, numberOfThreads, sharedMemorySize>>>(cudaMatrixA, cudaMatrixB, cudaResultMatrix, matrixWidth);
		// 			} else {
		// 				int sharedMemorySize = sizeOfMatrix*sizeof(int);
		// 				hipEventRecord(start);
		// 				cout << "CACHING FIRST MATRIX:" << endl;
		// 				min_plus_kernel_cache_first<<<numberOfThreadBlocks, numberOfThreads, sharedMemorySize>>>(cudaMatrixA, cudaMatrixB, cudaResultMatrix, matrixWidth);
		// 			}
		
		// 		} else {
		// 			cout << "CACHING ROW IN MATRIX:" << endl;
		// 			int numberOfThreads = min(matrixWidth, 1024);
		// 			int numberOfThreadBlocks = sizeOfMatrix/numberOfThreads;
		// 			int sharedMemorySize = matrixWidth*sizeof(int);
		// 			hipEventRecord(start);
		// 			min_plus<<<numberOfThreadBlocks, numberOfThreads, sharedMemorySize>>>(cudaMatrixA, cudaMatrixB, cudaResultMatrix, matrixWidth);
		// 		}
		// 		for (int j = 0; j < sizeOfMatrix; j++) {
		// 			cudaMatrixB[j] = cudaResultMatrix[j];
		// 			cudaResultMatrix[j] = INT_MAX;
		// 		}
		// 	} else {
		// 		if (matrixWidth < 128) {
		// 			int numberOfThreadBlocks = ceil(sizeOfMatrix/1024.0);
		// 			int numberOfThreads = min(1024, sizeOfMatrix);
		
		// 			if (matrixWidth < 64) {
		// 				int sharedMemorySize = sizeOfMatrix*2*sizeof(int);
		// 				hipEventRecord(start);
		// 				cout << "CACHING BOTH MATRICES:" << endl;
		// 				min_plus_kernel_cache_both<<<numberOfThreadBlocks, numberOfThreads, sharedMemorySize>>>(cudaMatrixA, cudaMatrixB, cudaResultMatrix, matrixWidth);
		// 			} else {
		// 				int sharedMemorySize = sizeOfMatrix*sizeof(int);
		// 				hipEventRecord(start);
		// 				cout << "CACHING FIRST MATRIX:" << endl;
		// 				min_plus_kernel_cache_first<<<numberOfThreadBlocks, numberOfThreads, sharedMemorySize>>>(cudaMatrixA, cudaMatrixB, cudaResultMatrix, matrixWidth);
		// 			}
		
		// 		} else {
		// 			cout << "CACHING ROW IN MATRIX:" << endl;
		// 			int numberOfThreads = min(matrixWidth, 1024);
		// 			int numberOfThreadBlocks = sizeOfMatrix/numberOfThreads;
		// 			int sharedMemorySize = matrixWidth*sizeof(int);
		// 			hipEventRecord(start);
		// 			min_plus<<<numberOfThreadBlocks, numberOfThreads, sharedMemorySize>>>(cudaMatrixA, cudaMatrixB, cudaResultMatrix, matrixWidth);
		// 		}
		// 	}
		// }

		// INITIAL KERNEL
		if (matrixWidth < 128) {
			int numberOfThreadBlocks = ceil(sizeOfMatrix/1024.0);
			int numberOfThreads = min(1024, sizeOfMatrix);

			if (matrixWidth < 64) {
				int sharedMemorySize = sizeOfMatrix*2*sizeof(int);
				hipEventRecord(start);
				cout << "CACHING BOTH MATRICES:" << endl;
				min_plus_kernel_cache_both<<<numberOfThreadBlocks, numberOfThreads, sharedMemorySize>>>(cudaMatrixA, cudaMatrixB, cudaResultMatrix, matrixWidth);
			} else {
				int sharedMemorySize = sizeOfMatrix*sizeof(int);
				hipEventRecord(start);
				cout << "CACHING FIRST MATRIX:" << endl;
				min_plus_kernel_cache_first<<<numberOfThreadBlocks, numberOfThreads, sharedMemorySize>>>(cudaMatrixA, cudaMatrixB, cudaResultMatrix, matrixWidth);
			}

		} else {
			cout << "CACHING ROW IN MATRIX:" << endl;
			int numberOfThreads = min(matrixWidth, 1024);
			int numberOfThreadBlocks = sizeOfMatrix/numberOfThreads;
			int sharedMemorySize = matrixWidth*sizeof(int);
			hipEventRecord(start);
			min_plus<<<numberOfThreadBlocks, numberOfThreads, sharedMemorySize>>>(cudaMatrixA, cudaMatrixB, cudaResultMatrix, matrixWidth);
		}
		
		hipDeviceSynchronize();	
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventSynchronize(stop);

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		hipMemcpy(ResultMatrix, cudaResultMatrix, sizeof(int)*sizeOfMatrix, hipMemcpyDeviceToHost);		


		auto begin = chrono::high_resolution_clock::now();
		min_plus_serial(MatrixA, MatrixB, serialResultMatrix, matrixWidth);	
		auto end = chrono::high_resolution_clock::now();
		auto dur = end - begin;
		auto serialTime = chrono::duration_cast<chrono::milliseconds>(dur).count();

		// //validate ResultMatrix
		// bool ifEquiv = true;
		// for (int k = 0; k < sizeOfMatrix; k++) {
		// 	if (expected[k] != ResultMatrix[k]) {
		// 		ifEquiv = false;
		// 		break;
		// 	}
		// }
		bool check = equivChecker(expected,ResultMatrix, sizeOfMatrix);

		hipFree(cudaMatrixA);
		hipFree(cudaMatrixB);
		hipFree(cudaResultMatrix);

		if (check) {
			cout << "Computed min-plus multiplication for " << argv[i] << " correctly in " << milliseconds << " ms in parallel and " << serialTime << " milliseconds in serial." << endl;
		} else {
			for (int k = 0; k < sizeOfMatrix; k++) {
				if (k % matrixWidth == 0) {
					cout << endl;
				}

				cout << ResultMatrix[k] << " ";
			}

			cout << "Error computing min-plus for " << argv[i] << endl;
			//cout << endl << hipGetErrorString(hipGetLastError()) << endl;
			//hipError_t error = hipGetLastError();
		//	cout << hipGetLastError() << endl;
		}
	}
}


int main(int argc, char *argv[]) {
	implementAlgorithm(argc, argv);
	return 0;
}
