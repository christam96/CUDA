
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <climits>
#include <algorithm>

using namespace std;

__global__ void min_plus(int *matrix1, int *matrix2, int *result, int matrixWidth, size_t n) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("INDEX: " + index);

	int col = index % matrixWidth;
	int row = index/matrixWidth;

	for (int k = 0; k < matrixWidth; k++) {
		int firstNum = matrix1[row*matrixWidth + k];
		int secondNum = matrix2[k*matrixWidth + col];
		result[index] = min(result[index], firstNum + secondNum);
	}

}


int main() {

	const int matrixWidth = 4;
	const int n = matrixWidth * matrixWidth;    
	
	int matrix1[n] = {0, 62, 51, 77, 66, 0, 9, 96, 37, 53, 0, 60, 83, 25, 16, 0};
	int matrix2[n] = {0, 62, 51, 77, 66, 0, 9, 96, 37, 53, 0, 60, 83, 25, 16, 0};
	int result[n] = {0};



	for (int i = 0; i < n; ++i) {
		result[i] = INT_MAX;
	}

	
	int *matrix1d, *matrix2d, *resultd;
	hipMalloc((void **)&matrix1d, sizeof(int)*n);
	hipMalloc((void **)&matrix2d, sizeof(int)*n);
	hipMalloc((void **)&resultd, sizeof(int)*n);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(matrix1d, matrix1, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(matrix2d, matrix2, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(resultd, result, sizeof(int)*n, hipMemcpyHostToDevice);

	hipEventRecord(start);

	min_plus<<<1, 16>>>(matrix1d, matrix2d, resultd, matrixWidth, n);

	hipEventRecord(stop);

	hipMemcpy(result, resultd, sizeof(int)*n, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << milliseconds << " milliseconds" << endl;

	for (int i = 0; i < matrixWidth*matrixWidth; i++) {

		if (i % matrixWidth == 0){
			cout << endl;
		}
		cout << result[i] << " ";

	}
	
	hipFree(matrix1d);
	hipFree(matrix2d);
	hipFree(resultd);
	
	return 0;
}
